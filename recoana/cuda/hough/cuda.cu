#include "hip/hip_runtime.h"
// cuda.cu
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include "common.h"

static void HandleError( hipError_t err, const char *file, int line ) {
  if (err != hipSuccess) {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
    exit( EXIT_FAILURE );
  }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

data_t d_data;

__global__ void
hough_init_kernel(data_t data)
{
  int size = data.bins.x * data.bins.y * data.bins.r * data.bins.t;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= size) return;

  int ix = (tid % data.bins.x);
  int iy = (tid / data.bins.x) % data.bins.y;
  int ir = (tid / (data.bins.x * data.bins.y)) % data.bins.r;
  int it = (tid / (data.bins.x * data.bins.y * data.bins.r)) % data.bins.t;

  data.map.x[tid] = data.min.x + (0.5 + ix) * (data.max.x - data.min.x) / data.bins.x;
  data.map.y[tid] = data.min.y + (0.5 + iy) * (data.max.y - data.min.y) / data.bins.y;
  data.map.r[tid] = data.min.r + (0.5 + ir) * (data.max.r - data.min.r) / data.bins.r;
  data.map.t[tid] = data.min.t + (0.5 + it) * (data.max.t - data.min.t) / data.bins.t;
  
}

__global__ void
hough_transform_kernel(data_t data)
{
  int size = data.bins.x * data.bins.y * data.bins.r * data.bins.t;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= size) return;

  float cx = data.map.x[tid];
  float cy = data.map.y[tid];
  float cr = data.map.r[tid];
  float ct = data.map.t[tid];

  float ts1 = 1. / (data.sigma.t * sqrtf(2. * M_PI));
  float ts2 = -0.5 / (data.sigma.t * data.sigma.t);
  
  data.hough.h[tid] = 0.;
  for (int i = 0; i < data.points.n; ++i) {
    float dx = data.points.x[i] - cx;
    float dy = data.points.y[i] - cy;
    float dt = data.points.t[i] - ct;
    float dr = hypotf(dx, dy) - cr;
    float w = 0.11398351 * expf(-0.040816327 * dr * dr  ); // sigma = 3.5 
    float wt = ts1 * expf(ts2 * dt * dt  );
    data.hough.h[tid] += (w * wt);
  }
  
}

__global__ void
find_max_kernel(data_t data)
{
  __shared__ float shm[256];
  __shared__ int shmi[256];
  
  int size = data.bins.x * data.bins.y * data.bins.r * data.bins.t;
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= size) return;

  int tid = threadIdx.x;
  int bid = blockIdx.x;

  shm[tid] = data.hough.h[gid];
  shmi[tid] = gid;
  __syncthreads();

  for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
    if (tid < stride) {
      if (shm[tid + stride] > shm[tid]) {
	shm[tid] = shm[tid + stride];
	shmi[tid] = shmi[tid + stride];
      }
    }
    __syncthreads();
  }

  if (tid == 0) {
    data.hough.rh[bid] = shm[0];
    data.hough.rhi[bid] = shmi[0];
  }

}

void
hough_init(data_t h_data)
{
  int size = h_data.bins.x * h_data.bins.y * h_data.bins.r * h_data.bins.t;
  int grid_size = 1 + (size - 1) / 256;
  
  /** alloc device memory for data map **/
  HANDLE_ERROR( hipMalloc((void **)&d_data.map.x, size * sizeof(float)) );
  HANDLE_ERROR( hipMalloc((void **)&d_data.map.y, size * sizeof(float)) );
  HANDLE_ERROR( hipMalloc((void **)&d_data.map.r, size * sizeof(float)) );
  HANDLE_ERROR( hipMalloc((void **)&d_data.map.t, size * sizeof(float)) );
  
  /** launch init kernel to populate data map **/
  d_data.min = h_data.min;
  d_data.max = h_data.max;
  d_data.bins = h_data.bins;
  hough_init_kernel<<<grid_size, 256>>>(d_data);

  /** copy data map from device **/
  HANDLE_ERROR( hipMemcpy(h_data.map.x, d_data.map.x, size * sizeof(float), hipMemcpyDeviceToHost) );
  HANDLE_ERROR( hipMemcpy(h_data.map.y, d_data.map.y, size * sizeof(float), hipMemcpyDeviceToHost) );
  HANDLE_ERROR( hipMemcpy(h_data.map.r, d_data.map.r, size * sizeof(float), hipMemcpyDeviceToHost) );
  HANDLE_ERROR( hipMemcpy(h_data.map.t, d_data.map.t, size * sizeof(float), hipMemcpyDeviceToHost) );
  
  /** alloc device memory for data points **/
  HANDLE_ERROR( hipMalloc((void **)&d_data.points.x, 1024 * sizeof(float)) );
  HANDLE_ERROR( hipMalloc((void **)&d_data.points.y, 1024 * sizeof(float)) );
  HANDLE_ERROR( hipMalloc((void **)&d_data.points.t, 1024 * sizeof(float)) );
  
  HANDLE_ERROR( hipMalloc((void **)&d_data.hough.h, size * sizeof(float)) );
  HANDLE_ERROR( hipMalloc((void **)&d_data.hough.rh, grid_size * sizeof(float)) );
  HANDLE_ERROR( hipMalloc((void **)&d_data.hough.rhi, grid_size * sizeof(int)) );

}

void
hough_free()
{

  /** free device memory for data map **/
  hipFree(d_data.map.x);
  hipFree(d_data.map.y);
  hipFree(d_data.map.r);
  hipFree(d_data.map.t);

  /** free device memory for data points **/
  hipFree(d_data.points.x);
  hipFree(d_data.points.y);
  hipFree(d_data.points.t);
  
  /** free device memory for data hough **/
  hipFree(d_data.hough.h);
  hipFree(d_data.hough.rh);
  hipFree(d_data.hough.rhi);
  
}

void
hough_transform(data_t h_data)
{
  int size = h_data.bins.x * h_data.bins.y * h_data.bins.r * h_data.bins.t;
  int grid_size = 1 + (size - 1) / 256;

  /** copy data points to device **/
  HANDLE_ERROR( hipMemcpy(d_data.points.x, h_data.points.x, h_data.points.n * sizeof(float), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(d_data.points.y, h_data.points.y, h_data.points.n * sizeof(float), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(d_data.points.t, h_data.points.t, h_data.points.n * sizeof(float), hipMemcpyHostToDevice) );
  
  // launch kernel
  d_data.points.n = h_data.points.n;
  d_data.sigma.t = h_data.sigma.t;
  hough_transform_kernel<<<grid_size, 256>>>(d_data);
  find_max_kernel<<<grid_size, 256>>>(d_data);
  
  // copy data from device
  HANDLE_ERROR( hipMemcpy(h_data.hough.rh, d_data.hough.rh, grid_size * sizeof(float), hipMemcpyDeviceToHost) );
  HANDLE_ERROR( hipMemcpy(h_data.hough.rhi, d_data.hough.rhi, grid_size * sizeof(int), hipMemcpyDeviceToHost) );
}


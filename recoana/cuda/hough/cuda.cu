// cuda.cu
#include <hip/hip_runtime.h>

/** 
    Hough space GPU model
    the basic computational unit is a 10x10x10 block in the (x,y,r) space
    the lattice is defined by the cell spacing (dx,dy,dr)

    the full Hough space can be mapped with Nx,Ny,Nr blocks

    for example, if we want to map a Hough space 
    x = [-64, 64]
    y = [-64, 64]
    r = [ 32, 96]
    with a cell spacing of (1,1,1) we need (Nx,Ny,Nr) = (16,16,8) blocks
    Nx = 128 / dx / 8 = 16
    Ny = 128 / dy / 8 = 16
    Nr =  64 / dr / 8 = 8

**/

float *gpu_x = nullptr;
float *gpu_y = nullptr;
float *gpu_h = nullptr;

float *gpu_xmap = nullptr;
float *gpu_ymap = nullptr;
float *gpu_rmap = nullptr;

const float x_min = -9.75;
const float x_stp = 0.5;
const float y_min = -9.75;
const float y_stp = 0.5;
const float r_min = 30.;
const float r_stp = 0.5;

__global__ void
hough_gpu_init(float *xmap, float *ymap, float *rmap, int Nx, int Ny, int Nr) {

  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  int ix = threadIdx.x % 10;
  int iy = (threadIdx.x / 10) % 10;
  int ir = threadIdx.x / 100;
  
  int iX = blockIdx.x % Nx;
  int iY = (blockIdx.x / Nx) % Ny;
  int iR = blockIdx.x / (Nx * Ny);
  
  ix += iX * 10;
  iy += iY * 10;
  ir += iR * 10;
  
  float x = x_min + ix * x_stp;
  float y = y_min + iy * y_stp;
  float r = r_min + ir * r_stp;

  xmap[tid] = x;
  ymap[tid] = y;
  rmap[tid] = r;
  
}

__global__ void
hough_gpu_transform(float *xmap, float *ymap, float *rmap, float *x, float *y, float *h, int n) {

  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  float cx = xmap[tid];
  float cy = ymap[tid];
  float cr = rmap[tid];

  h[tid] = 0.;
  for (int i = 0; i < n; ++i) {
    float dx = cx - x[i];
    float dy = cy - y[i];
    float dr = hypotf(dx, dy) - cr;
    float w = 0.11398351 * expf(-0.040816327 * dr * dr  );
    h[tid] += w;
  }
  
}

void
hough_init(float *cpu_xmap, float *cpu_ymap, float *cpu_rmap, int Nx, int Ny, int Nr)
{
  int Nh = 1000 * Nx * Ny * Nr;
  
  // alloc device memory
  hipMalloc((void **)&gpu_x, 1024 * sizeof(float));
  hipMalloc((void **)&gpu_y, 1024 * sizeof(float));
  hipMalloc((void **)&gpu_h, Nh * sizeof(float));

  hipMalloc((void **)&gpu_xmap, Nh * sizeof(float));
  hipMalloc((void **)&gpu_ymap, Nh * sizeof(float));
  hipMalloc((void **)&gpu_rmap, Nh * sizeof(float));

  // launch kernel
  dim3 block_size(1000, 1, 1);
  dim3 grid_size(Nx * Ny * Nr, 1, 1);
  hough_gpu_init<<<grid_size, block_size>>>(gpu_xmap, gpu_ymap, gpu_rmap, Nx, Ny, Nr);

  // copy data from device
  hipMemcpy(cpu_xmap, gpu_xmap, Nh * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(cpu_ymap, gpu_ymap, Nh * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(cpu_rmap, gpu_rmap, Nh * sizeof(float), hipMemcpyDeviceToHost);

}

void
hough_free()
{
  // free device memory
  hipFree(gpu_x);
  hipFree(gpu_y);
  hipFree(gpu_h);
  
  hipFree(gpu_xmap);
  hipFree(gpu_ymap);
  hipFree(gpu_rmap);
}

void
hough_transform(float *cpu_x, float *cpu_y, float *cpu_h, int cpu_n, int Nx, int Ny, int Nr)
{
  int Nh = 1000 * Nx * Ny * Nr;

  // copy data to device
  hipMemcpy(gpu_x, cpu_x, cpu_n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(gpu_y, cpu_y, cpu_n * sizeof(float), hipMemcpyHostToDevice);
  
  // launch kernel
  dim3 block_size(1000, 1, 1);
  dim3 grid_size(Nx * Ny * Nr, 1, 1);
  hough_gpu_transform<<<grid_size, block_size>>>(gpu_xmap, gpu_ymap, gpu_rmap, gpu_x, gpu_y, gpu_h, cpu_n);

  // copy data from device
  hipMemcpy(cpu_h, gpu_h, Nh * sizeof(float), hipMemcpyDeviceToHost);
}

